#include "hip/hip_runtime.h"
// Author: Dai-Ni Hsieh (dnhsieh@jhu.edu)
// Date  : 11/17/2020

#include <cmath>
#include "besselk.h"
#include "polybesselk.h"
#include "matvec.h"
#include "constants.h"

inline void setBesselkCoefficients()
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_P01Vec), P01Vec, sizeof(double) * (P01Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q01Vec), Q01Vec, sizeof(double) * (Q01Deg + 1), 0, hipMemcpyHostToDevice);	

	hipMemcpyToSymbol(HIP_SYMBOL(c_P02Vec), P02Vec, sizeof(double) * (P02Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q02Vec), Q02Vec, sizeof(double) * (Q02Deg + 1), 0, hipMemcpyHostToDevice);	

	hipMemcpyToSymbol(HIP_SYMBOL(c_P03Vec), P03Vec, sizeof(double) * (P03Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q03Vec), Q03Vec, sizeof(double) * (Q03Deg + 1), 0, hipMemcpyHostToDevice);	

	hipMemcpyToSymbol(HIP_SYMBOL(c_P11Vec), P11Vec, sizeof(double) * (P11Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q11Vec), Q11Vec, sizeof(double) * (Q11Deg + 1), 0, hipMemcpyHostToDevice);	

	hipMemcpyToSymbol(HIP_SYMBOL(c_P12Vec), P12Vec, sizeof(double) * (P12Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q12Vec), Q12Vec, sizeof(double) * (Q12Deg + 1), 0, hipMemcpyHostToDevice);	

	hipMemcpyToSymbol(HIP_SYMBOL(c_P13Vec), P13Vec, sizeof(double) * (P13Deg + 1), 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(c_Q13Vec), Q13Vec, sizeof(double) * (Q13Deg + 1), 0, hipMemcpyHostToDevice);	

	return;
}

__global__ void gaussian(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijSqu = eucdistSqu(qiVec, qjVec) / (knlWidth * knlWidth);
		double knlVal = exp(-dijSqu);

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void gaussian(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                         double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijSqu = eucdistSqu(qiVec, qjVec) / (knlWidth * knlWidth);
		double knlVal = exp(-dijSqu);

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern0(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;

		double p1Val;
		p1Fcn(p1Val, dijVal);

		double knlVal = p1Val;

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern0(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                        double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;

		double p1Val;
		p1Fcn(p1Val, dijVal);

		double knlVal = p1Val;

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern1(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = 0.5 * (p0Val + 2.0 * p1Val);

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern1(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                        double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = 0.5 * (p0Val + 2.0 * p1Val);

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern2(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = (4.0 * p0Val + (8.0 + dijSqu) * p1Val) / 8.0;

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern2(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                        double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = (4.0 * p0Val + (8.0 + dijSqu) * p1Val) / 8.0;

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern3(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = ((24.0 + dijSqu) * p0Val + 8.0 * (6.0 + dijSqu) * p1Val) / 48.0;

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern3(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                        double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = ((24.0 + dijSqu) * p0Val + 8.0 * (6.0 + dijSqu) * p1Val) / 48.0;

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern4(double *d_knlMat, double *d_lmkMat, double knlWidth, int lmkNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkNum && colIdx < lmkNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkMat, rowIdx, lmkNum);
		getVector(qjVec, d_lmkMat, colIdx, lmkNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = (12.0 * (16.0 + dijSqu) * p0Val + (384.0 + dijSqu * (72.0 + dijSqu)) * p1Val) / 384.0;

		d_knlMat[colIdx * lmkNum + rowIdx] = knlVal;
	}

	return;
}

__global__ void matern4(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat,
                        double knlWidth, int lmkiNum, int lmkjNum)
{
	int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
	if ( rowIdx < lmkiNum && colIdx < lmkjNum )
	{
		vector qiVec, qjVec;
		getVector(qiVec, d_lmkiMat, rowIdx, lmkiNum);
		getVector(qjVec, d_lmkjMat, colIdx, lmkjNum);

		double dijVal = eucdist(qiVec, qjVec) / knlWidth;
		double dijSqu = dijVal * dijVal;

		double p0Val, p1Val;
		p0Fcn(p0Val, dijVal);
		p1Fcn(p1Val, dijVal);

		double knlVal = (12.0 * (16.0 + dijSqu) * p0Val + (384.0 + dijSqu * (72.0 + dijSqu)) * p1Val) / 384.0;

		d_knlMat[colIdx * lmkiNum + rowIdx] = knlVal;
	}

	return;
}

void computeKernel(double *d_knlMat, double *d_lmkMat, int knlOrder, double knlWidth, int lmkNum)
{
	// order 0 to 4: Matern kernel of order 0 to 4
	// order     -1: Gaussian kernel

	setBesselkCoefficients();

	int  gridRow = (lmkNum - 1) / BLKROW + 1;
	dim3 blkNum(gridRow, gridRow);
	dim3 blkDim( BLKROW,  BLKROW);

	switch ( knlOrder )
	{
		case -1:
			gaussian <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;

		case 0:
			matern0 <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;

		case 1:
			matern1 <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;

		case 2:
			matern2 <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;

		case 3:
			matern3 <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;

		case 4:
			matern4 <<<blkNum, blkDim>>> (d_knlMat, d_lmkMat, knlWidth, lmkNum);
			break;
	}

	return;
}

void computeKernel(double *d_knlMat, double *d_lmkiMat, double *d_lmkjMat, 
                   int knlOrder, double knlWidth, int lmkiNum, int lmkjNum)
{
	// order 0 to 4: Matern kernel of order 0 to 4
	// order     -1: Gaussian kernel

	setBesselkCoefficients();

	int  gridRow = (lmkiNum - 1) / BLKROW + 1;
	int  gridCol = (lmkjNum - 1) / BLKROW + 1;
	dim3 blkNum(gridRow, gridCol);
	dim3 blkDim( BLKROW,  BLKROW);

	switch ( knlOrder )
	{
		case -1:
			gaussian <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;

		case 0:
			matern0 <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;

		case 1:
			matern1 <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;

		case 2:
			matern2 <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;

		case 3:
			matern3 <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;

		case 4:
			matern4 <<<blkNum, blkDim>>> (d_knlMat, d_lmkiMat, d_lmkjMat, knlWidth, lmkiNum, lmkjNum);
			break;
	}

	return;
}

